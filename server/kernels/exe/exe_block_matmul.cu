#include "hip/hip_runtime.h"
// native_cuda_block_matmul.cu
// Build: nvcc -O3 -std=c++17 -arch=sm_70 -o exe-block-matmul-cuda native_cuda_block_matmul.cu

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>
#include <vector>
#include <cstdlib>
#include <iostream>

using i32 = int32_t;

static void die(const char* msg){
    std::fprintf(stderr, "%s\n", msg);
    std::exit(1);
}

static void read_all_stdin(std::vector<uint8_t>& buffer) {
    const size_t CHUNK_SIZE = 1 << 20; // 1MB chunks
    uint8_t chunk[CHUNK_SIZE];
    size_t bytes_read;

    while ((bytes_read = std::fread(chunk, 1, CHUNK_SIZE, stdin)) > 0) {
        buffer.insert(buffer.end(), chunk, chunk + bytes_read);
    }

    if (std::ferror(stdin)) {
        die("Error reading from stdin");
    }
}

static void write_exact(const void* src, size_t n){
    const uint8_t* p = static_cast<const uint8_t*>(src);
    size_t put = 0;
    while(put < n){
        size_t w = std::fwrite(p + put, 1, n - put, stdout);
        if(w == 0) die("write error to stdout");
        put += w;
    }
}

#ifndef TILE
#define TILE 16
#endif

__global__ void block_matmul_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int rows, int K, int cols)
{
    __shared__ float As[TILE][TILE];
    __shared__ float Bs[TILE][TILE];

    int c = blockIdx.x * TILE + threadIdx.x; // col in C
    int r = blockIdx.y * TILE + threadIdx.y; // row in C

    float acc = 0.0f;
    int tiles = (K + TILE - 1) / TILE;

    for(int t=0; t<tiles; ++t){
        int kx = t*TILE + threadIdx.x;
        int ky = t*TILE + threadIdx.y;

        As[threadIdx.y][threadIdx.x] = (r<rows && kx<K) ? A[r*K + kx] : 0.0f;
        Bs[threadIdx.y][threadIdx.x] = (c<cols && ky<K) ? B[ky*cols + c] : 0.0f;

        __syncthreads();

        #pragma unroll
        for(int k=0;k<TILE;k++){
            acc += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        __syncthreads();
    }

    if(r<rows && c<cols) C[r*cols + c] = acc;
}

int main(){
    // 1) uniforms
    i32 rows=0, K=0, cols=0;
    read_exact(&rows,4); read_exact(&K,4); read_exact(&cols,4);
    if(rows<=0 || K<=0 || cols<=0) die("invalid uniforms");

    // 2) inputs
    size_t aN = (size_t)rows*(size_t)K;
    size_t bN = (size_t)K*(size_t)cols;
    size_t cN = (size_t)rows*(size_t)cols;

    std::vector<float> A(aN), B(bN), C(cN);
    read_exact(A.data(), aN*sizeof(float));
    read_exact(B.data(), bN*sizeof(float));

    // 3) device
    float *dA=nullptr,*dB=nullptr,*dC=nullptr;
    hipMalloc(&dA, aN*sizeof(float));
    hipMalloc(&dB, bN*sizeof(float));
    hipMalloc(&dC, cN*sizeof(float));
    hipMemcpy(dA, A.data(), aN*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, B.data(), bN*sizeof(float), hipMemcpyHostToDevice);

    dim3 block(TILE, TILE, 1);
    dim3 grid( (cols+TILE-1)/TILE, (rows+TILE-1)/TILE, 1 );
    block_matmul_kernel<<<grid, block>>>(dA, dB, dC, rows, K, cols);
    hipError_t err = hipDeviceSynchronize();
    if(err != hipSuccess) {
        std::fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return 2;
    }

    hipMemcpy(C.data(), dC, cN*sizeof(float), hipMemcpyDeviceToHost);
    write_exact(C.data(), cN*sizeof(float));

    hipFree(dA); hipFree(dB); hipFree(dC);
    return 0;
}

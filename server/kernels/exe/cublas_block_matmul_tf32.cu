// cublas_block_matmul_tf32.cu
// Build (Ampere+ recommended): 
//   nvcc -O3 -std=c++17 -arch=sm_80 -lcublas -o exe-block-matmul-cublas-tf32 cublas_block_matmul_tf32.cu
//
// Behavior:
// - Reads M, K, N (int32) then A[M*K], B[K*N] as float32 from stdin (row-major).
// - Uses cuBLAS GEMM (float32) with TF32 Tensor Cores enabled to compute C = A @ B.
// - Writes C[M*N] as float32 to stdout (row-major).
//
// Notes:
// - Converts to column-major on device for cuBLAS.
// - Explicitly enables TF32 tensor-op math; on pre-Ampere GPUs, this is ignored.

#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>

using i32 = int32_t;

static void die(const char* msg){
    std::fprintf(stderr, "%s\n", msg);
    std::exit(1);
}

static void read_exact(void* dst, size_t n){
    size_t off = 0;
    while(off < n){
        size_t got = std::fread((char*)dst + off, 1, n - off, stdin);
        if(got == 0) die("read_exact: unexpected EOF");
        off += got;
    }
}

static void write_exact(const void* src, size_t n){
    size_t off = 0;
    while(off < n){
        size_t put = std::fwrite((char*)src + off, 1, n - off, stdout);
        if(put == 0) die("write_exact: short write");
        off += put;
    }
}

#define CUDA_CHECK(x) do { hipError_t e=(x); if(e!=hipSuccess){ \
    std::fprintf(stderr,"CUDA error %s:%d: %s\n",__FILE__,__LINE__,hipGetErrorString(e)); std::exit(2);} } while(0)

#define CUBLAS_CHECK(x) do { hipblasStatus_t s=(x); if(s!=HIPBLAS_STATUS_SUCCESS){ \
    std::fprintf(stderr,"cuBLAS error %s:%d: status=%d\n",__FILE__,__LINE__,(int)s); std::exit(3);} } while(0)

// Convert row-major float (rows x cols) -> column-major float
__global__ void row_to_col_f32(const float* __restrict__ src, float* __restrict__ dst, int rows, int cols){
    int i = blockIdx.y * blockDim.y + threadIdx.y; // row
    int j = blockIdx.x * blockDim.x + threadIdx.x; // col
    if(i < rows && j < cols){
        dst[i + j * rows] = src[i * cols + j];
    }
}

// Convert column-major float -> row-major float
__global__ void col_to_row_f32(const float* __restrict__ src, float* __restrict__ dst, int rows, int cols){
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < rows && j < cols){
        dst[i * cols + j] = src[i + j * rows];
    }
}

int main(){
    // Read dims
    i32 M=0,K=0,N=0;
    read_exact(&M,4); read_exact(&K,4); read_exact(&N,4);
    if(M<0||K<0||N<0) die("Negative dimensions are not allowed.");
    size_t aN = (size_t)M * (size_t)K;
    size_t bN = (size_t)K * (size_t)N;
    size_t cN = (size_t)M * (size_t)N;

    // Host buffers
    std::vector<float> A(aN), B(bN), C(cN, 0.0f);
    if(aN) read_exact(A.data(), aN*sizeof(float));
    if(bN) read_exact(B.data(), bN*sizeof(float));
    if(M==0 || K==0 || N==0){
        if(cN) write_exact(C.data(), cN*sizeof(float));
        return 0;
    }

    // Device buffers
    float *dA_row=nullptr, *dB_row=nullptr;
    float *dA_col=nullptr, *dB_col=nullptr, *dC_col=nullptr;
    CUDA_CHECK(hipMalloc(&dA_row, aN*sizeof(float)));
    CUDA_CHECK(hipMalloc(&dB_row, bN*sizeof(float)));
    CUDA_CHECK(hipMemcpy(dA_row, A.data(), aN*sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB_row, B.data(), bN*sizeof(float), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&dA_col, (size_t)M*(size_t)K*sizeof(float)));
    CUDA_CHECK(hipMalloc(&dB_col, (size_t)K*(size_t)N*sizeof(float)));
    CUDA_CHECK(hipMalloc(&dC_col, (size_t)M*(size_t)N*sizeof(float)));

    dim3 block(32, 8);
    dim3 gridA((K+block.x-1)/block.x, (M+block.y-1)/block.y);
    dim3 gridB((N+block.x-1)/block.x, (K+block.y-1)/block.y);
    dim3 gridC((N+block.x-1)/block.x, (M+block.y-1)/block.y);
    row_to_col_f32<<<gridA, block>>>(dA_row, dA_col, M, K);
    row_to_col_f32<<<gridB, block>>>(dB_row, dB_col, K, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipFree(dA_row));
    CUDA_CHECK(hipFree(dB_row));

    // cuBLAS handle
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Enable TF32 Tensor Core math (ignored on pre-Ampere)
    #ifdef CUBLAS_TF32_TENSOR_OP_MATH
    CUBLAS_CHECK(cublasSetMathMode(handle, CUBLAS_TF32_TENSOR_OP_MATH));
    #else
    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
    #endif

    const float alpha = 1.0f, beta = 0.0f;
    // Column-major GEMM with TF32 acceleration
    CUBLAS_CHECK(hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        M, N, K,
        &alpha,
        dA_col, HIP_R_32F, M,
        dB_col, HIP_R_32F, K,
        &beta,
        dC_col, HIP_R_32F, M,
        HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    // Copy back as row-major
    float *dC_row=nullptr;
    CUDA_CHECK(hipMalloc(&dC_row, cN*sizeof(float)));
    col_to_row_f32<<<gridC, block>>>(dC_col, dC_row, M, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(C.data(), dC_row, cN*sizeof(float), hipMemcpyDeviceToHost));

    // cleanup
    hipblasDestroy(handle);
    CUDA_CHECK(hipFree(dA_col));
    CUDA_CHECK(hipFree(dB_col));
    CUDA_CHECK(hipFree(dC_col));
    CUDA_CHECK(hipFree(dC_row));

    // write
    write_exact(C.data(), cN*sizeof(float));
    return 0;
}

// cublas_block_matmul_bf16.cu
// Build (Ampere+ REQUIRED for BF16):
//   nvcc -O3 -std=c++17 -arch=sm_80 -lcublas -o exe-block-matmul-cublas-bf16 cublas_block_matmul_bf16.cu
//
// Behavior:
// - Reads M, K, N (int32) then A[M*K], B[K*N] as float32 from stdin (row-major).
// - Converts A, B to __nv_bfloat16 (column-major) on device, runs cuBLAS GEMM using BF16 Tensor Cores with FP32 accumulation.
// - Writes C[M*N] as float32 to stdout (row-major).

#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_bf16.h>

using i32 = int32_t;

static void die(const char* msg){
    std::fprintf(stderr, "%s\n", msg);
    std::exit(1);
}

static void read_exact(void* dst, size_t n){
    size_t off = 0;
    while(off < n){
        size_t got = std::fread((char*)dst + off, 1, n - off, stdin);
        if(got == 0) die("read_exact: unexpected EOF");
        off += got;
    }
}

static void write_exact(const void* src, size_t n){
    size_t off = 0;
    while(off < n){
        size_t put = std::fwrite((char*)src + off, 1, n - off, stdout);
        if(put == 0) die("write_exact: short write");
        off += put;
    }
}

#define CUDA_CHECK(x) do { hipError_t e=(x); if(e!=hipSuccess){ \
    std::fprintf(stderr,"CUDA error %s:%d: %s\n",__FILE__,__LINE__,hipGetErrorString(e)); std::exit(2);} } while(0)

#define CUBLAS_CHECK(x) do { hipblasStatus_t s=(x); if(s!=HIPBLAS_STATUS_SUCCESS){ \
    std::fprintf(stderr,"cuBLAS error %s:%d: status=%d\n",__FILE__,__LINE__,(int)s); std::exit(3);} } while(0)

// Convert row-major float (rows x cols) -> column-major __nv_bfloat16
__global__ void row_to_col_f32_to_bf16(const float* __restrict__ src, __hip_bfloat16* __restrict__ dst, int rows, int cols){
    int i = blockIdx.y * blockDim.y + threadIdx.y; // row
    int j = blockIdx.x * blockDim.x + threadIdx.x; // col
    if(i < rows && j < cols){
        float v = src[i * cols + j];
        dst[i + j * rows] = __float2bfloat16(v);
    }
}

// Convert column-major float -> row-major float
__global__ void col_to_row_f32(const float* __restrict__ src, float* __restrict__ dst, int rows, int cols){
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < rows && j < cols){
        dst[i * cols + j] = src[i + j * rows];
    }
}

int main(){
    // Read dims
    i32 M=0,K=0,N=0;
    read_exact(&M,4); read_exact(&K,4); read_exact(&N,4);
    if(M<0||K<0||N<0) die("Negative dimensions are not allowed.");
    size_t aN = (size_t)M * (size_t)K;
    size_t bN = (size_t)K * (size_t)N;
    size_t cN = (size_t)M * (size_t)N;

    // Host buffers
    std::vector<float> A(aN), B(bN), C(cN, 0.0f);
    if(aN) read_exact(A.data(), aN*sizeof(float));
    if(bN) read_exact(B.data(), bN*sizeof(float));
    if(M==0 || K==0 || N==0){
        if(cN) write_exact(C.data(), cN*sizeof(float));
        return 0;
    }

    // Device feature check: BF16 requires SM 8.0+
    hipDeviceProp_t prop{};
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    if(prop.major < 8){
        die("BF16 requires Ampere (SM80) or newer GPU.");
    }

    // Device buffers
    float *dA_row=nullptr, *dB_row=nullptr;
    __hip_bfloat16 *dA_col=nullptr, *dB_col=nullptr;
    float *dC_col=nullptr;
    CUDA_CHECK(hipMalloc(&dA_row, aN*sizeof(float)));
    CUDA_CHECK(hipMalloc(&dB_row, bN*sizeof(float)));
    CUDA_CHECK(hipMemcpy(dA_row, A.data(), aN*sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB_row, B.data(), bN*sizeof(float), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&dA_col, (size_t)M*(size_t)K*sizeof(__hip_bfloat16)));
    CUDA_CHECK(hipMalloc(&dB_col, (size_t)K*(size_t)N*sizeof(__hip_bfloat16)));
    CUDA_CHECK(hipMalloc(&dC_col, (size_t)M*(size_t)N*sizeof(float)));

    dim3 block(32, 8);
    dim3 gridA((K+block.x-1)/block.x, (M+block.y-1)/block.y);
    dim3 gridB((N+block.x-1)/block.x, (K+block.y-1)/block.y);
    dim3 gridC((N+block.x-1)/block.x, (M+block.y-1)/block.y);
    row_to_col_f32_to_bf16<<<gridA, block>>>(dA_row, dA_col, M, K);
    row_to_col_f32_to_bf16<<<gridB, block>>>(dB_row, dB_col, K, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipFree(dA_row));
    CUDA_CHECK(hipFree(dB_row));

    // cuBLAS
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    // Enable Tensor Ops (BF16 gemmEx requires tensor op algo)
    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

    const float alpha = 1.0f, beta = 0.0f;
    // GEMM: column-major: C(MxN) = A(MxK) * B(KxN)
    CUBLAS_CHECK(hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        M, N, K,
        &alpha,
        dA_col, HIP_R_16BF, M,
        dB_col, HIP_R_16BF, K,
        &beta,
        dC_col,  HIP_R_32F, M,
        HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    // Convert C (col-major float) back to row-major
    float *dC_row=nullptr;
    CUDA_CHECK(hipMalloc(&dC_row, cN*sizeof(float)));
    col_to_row_f32<<<gridC, block>>>(dC_col, dC_row, M, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(C.data(), dC_row, cN*sizeof(float), hipMemcpyDeviceToHost));

    // cleanup
    hipblasDestroy(handle);
    CUDA_CHECK(hipFree(dA_col));
    CUDA_CHECK(hipFree(dB_col));
    CUDA_CHECK(hipFree(dC_col));
    CUDA_CHECK(hipFree(dC_row));

    // write
    write_exact(C.data(), cN*sizeof(float));
    return 0;
}
